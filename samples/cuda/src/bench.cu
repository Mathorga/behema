#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <unistd.h>
#include <portia/portia.h>

int main(int argc, char **argv) {
    cortex_size_t cortex_width = 2500;
    cortex_size_t cortex_height = 1000;
    int32_t grid_size = (cortex_width * cortex_height) / BLOCK_SIZE;
    nh_radius_t nh_radius = 1;

    srand(time(NULL));

    error_code_t error;

    // Cortex init.
    cortex2d_t* even_cortex;
    cortex2d_t* odd_cortex;
    error = c2d_init(&even_cortex, cortex_width, cortex_height, nh_radius);
    error = c2d_init(&odd_cortex, cortex_width, cortex_height, nh_radius);
    c2d_copy(odd_cortex, even_cortex);

    // Copy cortexes to the device.
    cortex2d_t* d_even_cortex;
    cortex2d_t* d_odd_cortex;
    hipMalloc(&d_even_cortex, sizeof(cortex2d_t));
    cudaCheckError();
    hipMalloc(&d_odd_cortex, sizeof(cortex2d_t));
    cudaCheckError();
    error = c2d_to_device(d_even_cortex, even_cortex);
    error = c2d_to_device(d_odd_cortex, odd_cortex);

    // Input init.
    // input2d_t* host_input;
    // i2d_init(&host_input, (cortex_width / 2) - 10, 0, (cortex_width / 2) + 10, 1, DEFAULT_EXC_VALUE * 2, PULSE_MAPPING_FPROP);

    uint64_t start_time = millis();

    for (int i = 0; i < 1000; i++) {
        cortex2d_t* prev_cortex = i % 2 ? d_odd_cortex : d_even_cortex;
        cortex2d_t* next_cortex = i % 2 ? d_even_cortex : d_odd_cortex;

        // TODO Feed.millis

        c2d_tick<<<grid_size, BLOCK_SIZE>>>(prev_cortex, next_cortex);
        hipDeviceSynchronize();

        // usleep(100);
    }

    uint64_t end_time = millis();
    printf("\nCompleted 1000 iterations in %ldms\n", end_time - start_time);

    // Cleanup.
    c2d_destroy(even_cortex);
    c2d_destroy(odd_cortex);
    c2d_device_destroy(d_even_cortex);
    c2d_device_destroy(d_odd_cortex);

    return 0;
}